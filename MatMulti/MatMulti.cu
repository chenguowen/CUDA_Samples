#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <malloc.h>
#include <random>
#include <time.h>

const int threadPerBlock = 16;

texture<int> texA;
texture<int> texB;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

hipError_t mulWithCuda(const int *a, const int *b, int *result, const int M, const int N, const int S);

hipError_t mulWithCudaTex(const int *a, const int *b, int *result, const int M, const int N, const int S);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

/* MatMultiply：CPU下矩阵乘法
*  a:第一个矩阵指针，表示a[M][N];
*  b:第二个矩阵指针，表示b[N][S];
*  result:结果矩阵，表示为result[M][S];
*/
void CPUMatMultiply(const int * a,const int * b, int *result,const int M,const int N,const int S)
{
	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < S; j++)
		{
			int index = i * S + j;
			result[index] = 0;

			//计算每一个元素的结果
			for (int k = 0; k < N; k++)
			{
				result[index] += a[i * N + k] * b[k * S + j];
			}
		}
	}
}

/* gpuMatMultKernel：GPU下矩阵乘法核函数
*  a:第一个矩阵指针，表示a[M][N]
*  b:第二个矩阵指针，表示b[N][S]
*  result:结果矩阵，表示result[M][S]
*/
__global__ void gpuMatMultKernel(const int *a, const int *b, int *result, const int M, const int N, const int S)
{
	//int threadId = threadIdx.x + blockIdx.x * blockDim.x;

	int threadId = (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId < M * S)
	{
		int row = threadId / S;
		int column = threadId % S;

		result[threadId] = 0;
		for (int i = 0; i < N; i++)
		{
			result[threadId] += a[row * N + i] * b[i * S + column];
		}
	}
}

/* gpuMatMultWithSharedKernel：GPU下使用shared内存的矩阵乘法
*  a:第一个矩阵指针，表示a[height_A][width_A]
*  b:第二个矩阵指针，表示b[width_A][width_B]
*  result:结果矩阵，表示result[height_A][width_B]
*/
template<int BLOCK_SIZE>
__global__ void gpuMatMultWithSharedKernel(const int *a, const int *b, int *result, const int height_A, const int width_A, const int width_B)
{
	int block_x = blockIdx.x;
	int block_y = blockIdx.y;
	int thread_x = threadIdx.x;
	int thread_y = threadIdx.y;

	if ((thread_y + block_y * blockDim.y) * width_B + block_x * blockDim.x + thread_x >= height_A * width_B)
	{
		return;
	}

	const int begin_a = block_y * blockDim.y * width_A;
	const int end_a = begin_a + width_A - 1;
	const int step_a = blockDim.x;

	const int begin_b = block_x * blockDim.x;
	const int step_b = blockDim.y * width_B;

	int result_temp = 0;

	for (int index_a = begin_a, int index_b = begin_b;
		index_a < end_a; index_a += step_a, index_b += step_b)
	{
		__shared__ int SubMat_A[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ int SubMat_B[BLOCK_SIZE][BLOCK_SIZE];

		SubMat_A[thread_y][thread_x] = a[index_a + thread_y * width_A + thread_x];
		SubMat_B[thread_y][thread_x] = b[index_b + thread_y * width_B + thread_x];

		__syncthreads();

		for (int i = 0; i < BLOCK_SIZE; i++)
		{
			result_temp += SubMat_A[thread_y][i] * SubMat_B[i][thread_x];
		}

		__syncthreads();
	}

	int begin_result = block_y * blockDim.y * width_B + begin_b;
	result[begin_result + thread_y * width_B + thread_x] = result_temp;
}

/* gpuMatMultWithTextureKernel：GPU下使用texture内存的矩阵乘法
*  result：结果矩阵，表示为result[M][S];
*  M：表示为矩阵A与矩阵result的行数
*  N：表示矩阵A的列数，矩阵B的行数
*  S：表示矩阵B和矩阵result的列数
*/
__global__ void gpuMatMultWithTextureKernel(int * result, const int M, const int N, const int S)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	if (offset < M * S)
	{
		int a = 0, b = 0;
		int temp_result = 0;
		for (int i = 0; i < N; i++)
		{
			a = tex1Dfetch(texA, y * N + i);
			b = tex1Dfetch(texB, i * S + x);
			temp_result += a * b;
		}
		result[offset] = temp_result;
	}
}


// main主函数，分别运行CPU和GPU矩阵乘法函数，比较二者的运行时间
int main()
{

	//确定矩阵的大小
	int M = 0, N = 0, S = 0;
	printf("please input the value of M (Mat a's row):");
	scanf("%d", &M);
	printf("please input the value of N (Mat a's column and Mat b's row):");
	scanf("%d", &N);
	printf("please input the value of S (Mat b's column):");
	scanf("%d", &S);

	//分配矩阵空间
	int * a = (int *)malloc(M * N * sizeof(int));
	if (NULL == a)
	{
		printf("the malloc of Mat a is failed!\n");
		return 0;
	}
	int * b = (int *)malloc(N * S * sizeof(int));
	if (NULL == b)
	{
		printf("the malloc of Mat b is failed!\n");
		return 0;
	}
	//cpu与gpu的结果矩阵分别存放
	int * cpuResult = (int *)malloc(M * S * sizeof(int));
	if (NULL == cpuResult)
	{
		printf("the malloc of Mat cpuResult is failed!\n");
		return 0;
	}
	int * gpuResult = (int *)malloc(M * S * sizeof(int));
	if (NULL == cpuResult)
	{
		printf("the malloc of Mat gpuResult is failed!\n");
		return 0;
	}

	//生成矩阵数据
	printf("\nstart random the Mat a...\n");
	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < N; j++)
		{
			a[i * N + j] = rand() % 5;
		}
	}

	printf("\nstart random the Mat b...\n");
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < S; j++)
		{
			b[i * S + j] = rand() % 5;
		}
	}

	//统计CPU运行乘法的时间
	clock_t start, finish;
	double totalTime = 0.0;
	start = clock();

	//调用CPU矩阵乘法函数
	CPUMatMultiply(a, b, cpuResult, M, N, S);

	finish = clock();
	totalTime = (double)(finish - start) / CLOCKS_PER_SEC;
	printf("\nThe total time is %lf seconds!\n", totalTime);

	//调用GPU矩阵乘法函数
	hipError_t cudaStatus = mulWithCuda(a, b, gpuResult, M, N, S);
	//hipError_t cudaStatus = mulWithCudaTex(a, b, gpuResult, M, N, S);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "mulWithCuda failed!");
		return 0;
	}
	//打印结果矩阵result
	/*printf("\nthe result of CPU :\n");
	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < S; j++)
		{
			printf("%d\t", cpuResult[i * M + j]);
		}
		printf("\n");
	}

	printf("\nthe result of GPU :\n");
	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < S; j++)
		{
			printf("%d\t", gpuResult[i * M + j]);
		}
		printf("\n");
	}*/

	//确认CPU和GPU矩阵乘法结果是否相同，从而说明结果是否正确
	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < S; j++)
		{
			if (cpuResult[i * M + j] != gpuResult[i * M + j])
			{
				printf("the Result isn't equal!\n");
				return 0;
			}
		}
	}

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}

// 调用CUDA运行GPU矩阵乘法核函数
hipError_t mulWithCuda(const int *a, const int *b, int *result, const int M, const int N, const int S)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_result = 0;

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_a, M * N * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_a failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_b, N * S * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_b failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_result, M * S * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_result failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_a, a, M * N * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudamemcpy dev_a failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, N * S * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy dev_b failed!\n");
		goto Error;
	}

	hipEvent_t gpuStart, gpuFinish;
	float elapsedTime;
	hipEventCreate(&gpuStart);
	hipEventCreate(&gpuFinish);
	hipEventRecord(gpuStart, 0);

	/*const int THREADNUM = 256;
	const int BLOCKNUM = (M * S + 255) / 256;*/

	const int BLOCK_SIZE = 16;
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid((S + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);
	gpuMatMultKernel << <grid, block >> >(dev_a, dev_b, dev_result, M, N, S);
	//gpuMatMultWithSharedKernel<16> << <grid, block >> >(dev_a, dev_b, dev_result, M, N, S);

	hipEventRecord(gpuFinish, 0);
	hipEventSynchronize(gpuFinish);
	hipEventElapsedTime(&elapsedTime, gpuStart, gpuFinish);
	printf("\nThe runing time of GPU on Mat Multiply is %f seconds.\n", elapsedTime / 1000.0);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "MulKernel launch failed: %s!\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize return Error code %d after Kernel launched!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(result, dev_result, M * S * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy result failed!\n");
		goto Error;
	}

Error:
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_result);

	return cudaStatus;
}

//调用CUDA运行GPU矩阵乘法核函数
//将矩阵A与矩阵B绑定到纹理内存中
hipError_t mulWithCudaTex(const int *a, const int *b, int *result, const int M, const int N, const int S)
{
	int * dev_a = 0;
	int * dev_b = 0;
	int * dev_result = 0;

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed! Do you have a CUDA_capable GPU installed?\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_a, M * N * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_a failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_b, N * S * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_b failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_result, M * S * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_result failed!\n");
		goto Error;
	}

	hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
	cudaStatus = hipBindTexture(NULL, texA, dev_a, desc, M * N * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipBindTexture texA failed!\n");
		goto Error;
	}

	cudaStatus = hipBindTexture(NULL, texB, dev_b, desc, N * S * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipBindTexture texB failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_a, a, M * N * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudamemcpy dev_a failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, N * S * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy dev_b failed!\n");
		goto Error;
	}

	hipEvent_t gpuStart, gpuFinish;
	float elapsedTime;
	hipEventCreate(&gpuStart);
	hipEventCreate(&gpuFinish);
	hipEventRecord(gpuStart, 0);

	const int BLOCK_SIZE = 16;
	if ((M % BLOCK_SIZE != 0) && (S % BLOCK_SIZE != 0))
	{
		fprintf(stderr, "M or S can't be dividen by 16!\n");
		goto Error;
	}

	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(S / BLOCK_SIZE, M / BLOCK_SIZE);
	gpuMatMultWithTextureKernel << <grid, block >> >(dev_result, M, N, S);

	hipEventRecord(gpuFinish, 0);
	hipEventSynchronize(gpuFinish);
	hipEventElapsedTime(&elapsedTime, gpuStart, gpuFinish);
	printf("\nThe runing time of GPU on Mat Multiply is %f seconds.\n", elapsedTime / 1000.0);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "MulKernel launch failed: %s!\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize return Error code %d after Kernel launched!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(result, dev_result, M * S * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy result failed!\n");
		goto Error;
	}

Error:
	hipUnbindTexture(texA);
	hipUnbindTexture(texB);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_result);

	return cudaStatus;

}

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <malloc.h>
#include <random>
#include <time.h>

#define threadPerBlock 4

//typedef int test_type;
typedef float test_type;
//typedef double test_type;
//typedef long long test_type;
//typedef unsigned short test_type;
//typedef unsigned char test_type;

texture<int> texA;
texture<int> texB;

hipError_t addWithCuda(int *c, const int *mul_a, const int *mul_b, unsigned int size);
hipError_t mulWithCuda(const int *mul_a, const int *mul_b, int *result, const int M, const int N, const int S);
template< class T_type > hipError_t mulWithCuda_Shared_ATA(const T_type *mul_a, T_type *result, const int M, const int N);
template< class T_type > hipError_t mulWithCuda_Shared(const T_type *mul_a, const T_type *mul_b, T_type *result, const int M, const int N, const int S) ; 
// hipError_t mulWithCuda_Shared(const int *mul_a, const int *mul_b, int *result, const int M, const int N, const int S);
hipError_t mulWithCudaTex(const int *mul_a, const int *mul_b, int *result, const int M, const int N, const int S);

__global__ void addKernel(int *c, const int *mul_a, const int *mul_b)
{
	int i = threadIdx.x;
	c[i] = mul_a[i] + mul_b[i];
}

/* MatMultiply：CPU下矩阵乘法
*  mul_a:第一个矩阵指针，表示mul_a[M][N];
*  mul_b:第二个矩阵指针，表示mul_b[N][S];
*  result:结果矩阵，表示为result[M][S];
*/
template< class T_type > void CPUMatMultiply(const T_type * mul_a, const T_type * mul_b, T_type *result, const int M, const int N, const int S)
{
	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < S; j++)
		{
			int index = i * S + j;
			result[index] = 0; 
			//计算每一个元素的结果
			for (int k = 0; k < N; k++)
			{
				result[index] += mul_a[i * N + k] * mul_b[k * S + j];
			}
		}
	}
}

/* gpuMatMultKernel：GPU下矩阵乘法核函数
*  mul_a:第一个矩阵指针，表示mul_a[M][N]
*  mul_b:第二个矩阵指针，表示mul_b[N][S]
*  result:结果矩阵，表示result[M][S]
*/
__global__ void gpuMatMultKernel(const int *mul_a, const int *mul_b, int *result, const int M, const int N, const int S)
{
	//int threadId = threadIdx.x + blockIdx.x * blockDim.x;

	int threadId = (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId < M * S)
	{
		int row = threadId / S;
		int column = threadId % S;

		result[threadId] = 0;
		for (int i = 0; i < N; i++)
		{
			result[threadId] += mul_a[row * N + i] * mul_b[i * S + column];
		}
	}
}
// 
template<int BLOCK_SIZE, class T_type > 
__global__ void gpuMatMultWithSharedKernel(const T_type *mul_a, const T_type *mul_b, T_type *result, const int height_A, const int width_A, const int width_B)
{
	int block_x = blockIdx.x;
	int block_y = blockIdx.y;
	int thread_x = threadIdx.x;
	int thread_y = threadIdx.y;

	if ((thread_y + block_y * blockDim.y) * width_B + block_x * blockDim.x + thread_x >= height_A * width_B)
	{
		return;
	}

	const int begin_a = block_y * blockDim.y * width_A;
	const int end_a = begin_a + width_A ;
	const int step_a = blockDim.x;

	const int begin_b = block_x * blockDim.x;
	const int step_b = blockDim.y * width_B;

	T_type result_temp = 0;

	for (int index_a = begin_a, int index_b = begin_b; index_a < end_a; index_a += step_a, index_b += step_b)
	{
		__shared__ T_type SubMat_A[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ T_type SubMat_B[BLOCK_SIZE][BLOCK_SIZE];

		SubMat_A[thread_y][thread_x] = mul_a[index_a + thread_y * width_A + thread_x];
		SubMat_B[thread_y][thread_x] = mul_b[index_b + thread_y * width_B + thread_x];

		//SubMat_A[thread_y][thread_x] = tex1Dfetch(texA, index_a + thread_y * width_A + thread_x);
		//SubMat_B[thread_y][thread_x] = tex1Dfetch(texB, index_b + thread_y * width_B + thread_x);

		__syncthreads();

		for (int i = 0; i < BLOCK_SIZE; i++)
		{
			result_temp += SubMat_A[thread_y][i] * SubMat_B[i][thread_x];
		}

		__syncthreads();
	}

	int begin_result = block_y * blockDim.y * width_B + begin_b;
	result[begin_result + thread_y * width_B + thread_x] = result_temp;
} 
 
// 
template<int BLOCK_SIZE, class T_type > __global__ void gpuMatMultWithSharedKernel_ATA(const T_type *mul_a, T_type *result, const int height_A, const int width_A)
{
	int block_x = blockIdx.x;
	int block_y = blockIdx.y;
	int thread_x = threadIdx.x;
	int thread_y = threadIdx.y;
	// 
	if ( thread_y + block_y * blockDim.y >= height_A || block_x * blockDim.x + thread_x >= height_A * height_A )
	{
		return;
	}
	// 
	//if ((thread_y + block_y * blockDim.y) * width_B + block_x * blockDim.x + thread_x >= height_A * width_B)
	//{
	//	return;
	//}

	const int begin_a = block_y * blockDim.y * width_A; // 
	const int end_a   = begin_a + width_A - 1;
	const int step_a  = blockDim.x;
	// 
	const int begin_b = block_x * blockDim.y * width_A; // 
	const int step_b  = blockDim.x ;
	//
	T_type result_temp = 0; // 
	for (int index_a = begin_a, int index_b = begin_b; index_a < end_a; index_a += step_a, index_b += step_b)
	{
		__shared__ T_type SubMat_A[BLOCK_SIZE][BLOCK_SIZE];//
		__shared__ T_type SubMat_B[BLOCK_SIZE][BLOCK_SIZE];//
		SubMat_A[thread_y][thread_x] = mul_a[index_a + thread_y * width_A + thread_x];
		SubMat_B[thread_y][thread_x] = mul_a[index_b + thread_y * width_A + thread_x];
		//SubMat_A[thread_y][thread_x] = tex1Dfetch(texA, index_a + thread_y * width_A + thread_x);
		//SubMat_B[thread_y][thread_x] = tex1Dfetch(texB, index_b + thread_y * width_B + thread_x);
		__syncthreads(); //
		//SubMat_A[thread_y][i] * SubMat_B[i][thread_x];
		for (int i = 0; i < BLOCK_SIZE; i++)
		{
			result_temp += SubMat_A[i][thread_y] * SubMat_B[i][thread_x];
		}
		__syncthreads();
	}
	int begin_result = block_y * blockDim.y * height_A + begin_b;
	result[begin_result + thread_y * height_A + thread_x] = result_temp;
	// result[(thread_y + block_y * blockDim.y) * height_A + block_x * blockDim.x + thread_x] = result_temp;
}


/* gpuMatMultWithSharedKernel：GPU下使用shared内存的矩阵乘法
*  mul_a:第一个矩阵指针，表示mul_a[height_A][width_A]
*  mul_b:第二个矩阵指针，表示mul_b[width_A][width_B]
*  result:结果矩阵，表示result[height_A][width_B]
*/
//template<int BLOCK_SIZE, class T >
//__global__ void gpuMatMultWithSharedKernel(const int *mul_a, const int *mul_b, int *result, const int height_A, const int width_A, const int width_B)
//{
//	int block_x = blockIdx.x;
//	int block_y = blockIdx.y;
//	int thread_x = threadIdx.x;
//	int thread_y = threadIdx.y;
//
//	if ((thread_y + block_y * blockDim.y) * width_B + block_x * blockDim.x + thread_x >= height_A * width_B)
//	{
//		return;
//	}
//
//	const int begin_a = block_y * blockDim.y * width_A;
//	const int end_a   = begin_a + width_A - 1;
//	const int step_a  = blockDim.x;
//
//	const int begin_b = block_x * blockDim.x;
//	const int step_b  = blockDim.y * width_B;
//
//	int result_temp = 0;
//
//	for (int index_a = begin_a, int index_b = begin_b;	index_a < end_a; index_a += step_a, index_b += step_b)
//	{
//		__shared__ int SubMat_A[BLOCK_SIZE][BLOCK_SIZE];
//		__shared__ int SubMat_B[BLOCK_SIZE][BLOCK_SIZE];
//
//		SubMat_A[thread_y][thread_x] = mul_a[index_a + thread_y * width_A + thread_x];
//		SubMat_B[thread_y][thread_x] = mul_b[index_b + thread_y * width_B + thread_x];
//
//		//SubMat_A[thread_y][thread_x] = tex1Dfetch(texA, index_a + thread_y * width_A + thread_x);
//		//SubMat_B[thread_y][thread_x] = tex1Dfetch(texB, index_b + thread_y * width_B + thread_x);
//
//		__syncthreads();
//
//		for (int i = 0; i < BLOCK_SIZE; i++)
//		{
//			result_temp += SubMat_A[thread_y][i] * SubMat_B[i][thread_x];
//		}
//
//		__syncthreads();
//	}
//
//	int begin_result = block_y * blockDim.y * width_B + begin_b;
//	result[begin_result + thread_y * width_B + thread_x] = result_temp;
//}

/* gpuMatMultWithTextureKernel：GPU下使用texture内存的矩阵乘法
*  result：结果矩阵，表示为result[M][S];
*  M：表示为矩阵A与矩阵result的行数
*  N：表示矩阵A的列数，矩阵B的行数
*  S：表示矩阵B和矩阵result的列数
*/
__global__ void gpuMatMultWithTextureKernel(int * result, const int M, const int N, const int S)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	if (offset < M * S)
	{
		int mul_a = 0, mul_b = 0;
		int temp_result = 0;
		for (int i = 0; i < N; i++)
		{
			mul_a = tex1Dfetch(texA, y * N + i);
			mul_b = tex1Dfetch(texB, i * S + x);
			temp_result += mul_a * mul_b;
		}
		result[offset] = temp_result;
	}
}


// main主函数，分别运行CPU和GPU矩阵乘法函数，比较二者的运行时间
int main()
{ 
	//确定矩阵的大小
	int M = 0, N = 0, S = 0;
	printf("please input the value of M (Mat mul_a's row):");
	scanf("%d", &M);
	printf("please input the value of N (Mat mul_a's column and Mat mul_b's row):");
	scanf("%d", &N);
	printf("please input the value of S (Mat mul_b's column):");
	scanf("%d", &S);

	//分配矩阵空间
	test_type * mul_a = (test_type *)malloc(M * N * sizeof(test_type)); if (NULL == mul_a){ printf("the malloc of Mat mul_a is failed!\n");	return 0; }
	test_type * mul_b = (test_type *)malloc(N * S * sizeof(test_type)); if (NULL == mul_b){ printf("the malloc of Mat mul_b is failed!\n");	return 0; }
	//cpu与gpu的结果矩阵分别存放
	test_type * cpuResult = (test_type *)malloc(M * S * sizeof(test_type)); if (NULL == cpuResult){ printf("the malloc of Mat cpuResult is failed!\n");	return 0; }
	test_type * gpuResult = (test_type *)malloc(M * S * sizeof(test_type)); if (NULL == cpuResult){ printf("the malloc of Mat gpuResult is failed!\n");	return 0; }

	//生成矩阵数据
	printf("\nstart random the Mat mul_a...\n");
	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < N; j++)
		{
			mul_a[i * N + j] = rand() % 5; // 
			mul_b[j * M + i] = mul_a[i * N + j]; 
		}
	}

	//printf("\nstart random the Mat mul_b...\n");
	//for (int i = 0; i < N; i++)
	//{
	//	for (int j = 0; j < S; j++)
	//	{
	//		mul_b[i * S + j] = mul_a[i + j * M];// rand() % 5;
	//	}
	//}

	double result = 0; 
	printf("\n  ..\n");
	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < N; j++)
		{
			result += abs(mul_a[i * N + j] - mul_b[i + j * M]);
		}
	}

	std::cout << "result =" << result << std::endl;

	//统计CPU运行乘法的时间
	clock_t start, finish;
	double totalTime = 0.0;

	start = clock(); //调用CPU矩阵乘法函数
	CPUMatMultiply<test_type>(mul_a, mul_b, cpuResult, M, N, S);	finish = clock();
	totalTime = (double)(finish - start) / CLOCKS_PER_SEC;
	printf("\nThe total time is %lf seconds!\n", totalTime);



	hipError_t cudaStatus5 = mulWithCuda_Shared_ATA<test_type>(mul_a, gpuResult, M, N); 
	printf("\n GPU-ATA result!\n", totalTime);
	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < S; j++)
		{
			std::cout << gpuResult[i * M + j] << " ";
		}
		std::cout << std::endl;
	}
	hipError_t cudaStatus1 = mulWithCuda_Shared<test_type>(mul_a, mul_b, gpuResult, M, N, S);
	printf("\n GPU result!\n", totalTime);
	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < S; j++)
		{
			std::cout << gpuResult[i * M + j] << " "; 
		}
		std::cout << std::endl;
	}
	printf("\n CPU result!\n", totalTime);
	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < S; j++)
		{
			std::cout << cpuResult[i * M + j] << " "; 
		}
		std::cout << std::endl;
	}

	printf("\n A matrix !\n", totalTime);
	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < N; j++)
		{
			std::cout << mul_a[i * N + j] << " ";
		}
		std::cout << std::endl;
	}
	
	printf("\n B matrix !\n", totalTime);
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < S; j++)
		{
			std::cout << mul_b[i * S + j] << " ";
		}
		std::cout << std::endl;
	}
	////调用GPU矩阵乘法函数
	//hipError_t cudaStatus0 = mulWithCuda(mul_a, mul_b, gpuResult, M, N, S); 
	//printf(" \nComparing mulWithCuda amd mulWithCpu \n");
	//for (int i = 0; i < M; i++)
	//{
	//	for (int j = 0; j < S; j++)
	//	{
	//		if (cpuResult[i * M + j] != gpuResult[i * M + j])
	//		{
	//			printf("the Result isn't equal!\n");
	//			return 0;
	//		}
	//	}
	//}
	//printf("the Result is equal!\n\n");


	//hipError_t cudaStatus4 = mulWithCudaTex(mul_a, mul_b, gpuResult, M, N, S);  
	//printf(" \nComparing mulWithCudaTex amd mulWithCpu \n");
	//for (int i = 0; i < M; i++)
	//{
	//	for (int j = 0; j < S; j++)
	//	{
	//		if (cpuResult[i * M + j] != gpuResult[i * M + j])
	//		{
	//			printf("the Result isn't equal!\n");
	//			return 0;
	//		}
	//	}
	//}
	//printf("the Result is equal!\n\n");



	//hipError_t cudaStatus1 = mulWithCuda_Shared<test_type>(mul_a, mul_b, gpuResult, M, N, S);
	printf(" \nComparing mulWithCuda_Shared amd mulWithCpu \n");
	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < S; j++)
		{
			if (cpuResult[i * M + j] != gpuResult[i * M + j])
			{
				printf("the Result isn't equal!\n");
				return 0;
			}
		}
	}
	printf("the Result is equal!\n\n");


	//hipError_t cudaStatus5 = mulWithCuda_Shared_ATA<test_type>(mul_a, gpuResult, M, N);

	//printf(" \nComparing mulWithCuda_Shared amd mulWithCpu \n");
	//for (int i = 0; i < M; i++)
	//{
	//	for (int j = 0; j < S; j++)
	//	{
	//		if (cpuResult[i * M + j] != gpuResult[i * M + j])
	//		{
	//			printf("the Result isn't equal!\n");
	//			return 0;
	//		}
	//	}
	//}
	//printf("the Result is equal!\n\n");



	//打印结果矩阵result
	/*printf("\nthe result of CPU :\n");
	for (int i = 0; i < M; i++)
	{
	for (int j = 0; j < S; j++)
	{
	printf("%d\t", cpuResult[i * M + j]);
	}
	printf("\n");
	}
	printf("\nthe result of GPU :\n");
	for (int i = 0; i < M; i++)
	{
	for (int j = 0; j < S; j++)
	{
	printf("%d\t", gpuResult[i * M + j]);
	}
	printf("\n");
	}*/

	//确认CPU和GPU矩阵乘法结果是否相同，从而说明结果是否正确


	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *mul_a, const int *mul_b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on mul_a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have mul_a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, mul_a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, mul_b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch mul_a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> >(dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}

// 调用CUDA运行GPU矩阵乘法核函数
hipError_t mulWithCuda(const int *mul_a, const int *mul_b, int *result, const int M, const int N, const int S)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_result = 0;

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);

	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed! Do you have mul_a CUDA-capable GPU installed?\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_a, M * N * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_a failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_b, N * S * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_b failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_result, M * S * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_result failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_a, mul_a, M * N * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudamemcpy dev_a failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, mul_b, N * S * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy dev_b failed!\n");
		goto Error;
	}

	hipEvent_t gpuStart, gpuFinish;
	float elapsedTime;
	hipEventCreate(&gpuStart);
	hipEventCreate(&gpuFinish);
	hipEventRecord(gpuStart, 0);

	/*const int THREADNUM = 256;
	const int BLOCKNUM = (M * S + 255) / 256;*/

	const int BLOCK_SIZE = 16;
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid((S + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);
	// gpuMatMultKernel << <grid, block >> >(dev_a, dev_b, dev_result, M, N, S);
	gpuMatMultKernel << <grid, block >> >(dev_a, dev_b, dev_result, M, N, S);

	hipEventRecord(gpuFinish, 0);
	hipEventSynchronize(gpuFinish);
	hipEventElapsedTime(&elapsedTime, gpuStart, gpuFinish);

	printf("\nThe runing time of GPU on Mat Multiply using no texture is %f seconds.\n", elapsedTime / 1000.0);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "MulKernel launch failed: %s!\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize return Error code %d after Kernel launched!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(result, dev_result, M * S * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy result failed!\n");
		goto Error;
	}

Error:
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_result);
	return cudaStatus;
}
template< class T_type > hipError_t mulWithCuda_Shared_ATA(const T_type *mul_a, T_type *result, const int M, const int N)
{
	T_type *dev_a = 0;
	T_type *dev_result = 0;  
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0); if (cudaStatus != hipSuccess){ fprintf(stderr, "hipSetDevice failed! Do you have mul_a CUDA-capable GPU installed?\n");	goto Error; }
	cudaStatus = hipMalloc((void **)&dev_a, M * N * sizeof(T_type)); if (cudaStatus != hipSuccess){ fprintf(stderr, "hipMalloc dev_a failed!\n");	goto Error; }
	cudaStatus = hipMalloc((void **)&dev_result, M * M * sizeof(T_type));	if (cudaStatus != hipSuccess){ fprintf(stderr, "hipMalloc dev_result failed!\n");	goto Error; }
	cudaStatus = hipMemcpy(dev_a, mul_a, M * N * sizeof(T_type), hipMemcpyHostToDevice);	if (cudaStatus != hipSuccess){ fprintf(stderr, "cudamemcpy dev_a failed!\n");	goto Error; }
	hipEvent_t gpuStart, gpuFinish;  float elapsedTime;	hipEventCreate(&gpuStart);	hipEventCreate(&gpuFinish); hipEventRecord(gpuStart, 0);

	const int BLOCK_SIZE = threadPerBlock;
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid((M + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);
	gpuMatMultWithSharedKernel_ATA<threadPerBlock, T_type> << <grid, block >> >(dev_a, dev_result, M, N);

	hipEventRecord(gpuFinish, 0);
	hipEventSynchronize(gpuFinish);
	hipEventElapsedTime(&elapsedTime, gpuStart, gpuFinish);
	printf("\nThe runing time of GPU on Mat Multiply ATA using sharing memory is %f seconds.\n", elapsedTime / 1000.0);
	cudaStatus = hipGetLastError(); if (cudaStatus != hipSuccess)	{ fprintf(stderr, "MulKernel launch failed: %s!\n", hipGetErrorString(cudaStatus));	goto Error; }
	cudaStatus = hipMemcpy(result, dev_result, M * M * sizeof(T_type), hipMemcpyDeviceToHost); if (cudaStatus != hipSuccess){ fprintf(stderr, "hipMemcpy result failed!\n"); goto Error; }

Error:
	hipFree(dev_a);
	hipFree(dev_result); 
	return cudaStatus; 
}
// 调用CUDA运行GPU矩阵乘法核函数
template< class T_type > hipError_t mulWithCuda_Shared(const T_type *mul_a, const T_type *mul_b, T_type *result, const int M, const int N, const int S)
{
	T_type *dev_a = 0;
	T_type *dev_b = 0;
	T_type *dev_result = 0;

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0); if (cudaStatus != hipSuccess){fprintf(stderr, "hipSetDevice failed! Do you have mul_a CUDA-capable GPU installed?\n");	goto Error;}
	cudaStatus = hipMalloc((void **)&dev_a, M * N * sizeof(T_type)); if (cudaStatus != hipSuccess){	fprintf(stderr, "hipMalloc dev_a failed!\n");	goto Error;}
	cudaStatus = hipMalloc((void **)&dev_b, N * S * sizeof(T_type)); if (cudaStatus != hipSuccess){fprintf(stderr, "hipMalloc dev_b failed!\n");	goto Error;	}
	cudaStatus = hipMalloc((void **)&dev_result, M * S * sizeof(T_type));	if (cudaStatus != hipSuccess){	fprintf(stderr, "hipMalloc dev_result failed!\n");	goto Error;}

	//hipChannelFormatDesc desc = hipCreateChannelDesc<T_type>();
	//cudaStatus = hipBindTexture(NULL, texA, dev_a, desc, M * N * sizeof(T_type));	if (cudaStatus != hipSuccess){ fprintf(stderr, "hipBindTexture texA failed!\n");	goto Error; }
	//cudaStatus = hipBindTexture(NULL, texB, dev_b, desc, N * S * sizeof(T_type));	if (cudaStatus != hipSuccess){ fprintf(stderr, "hipBindTexture texB failed!\n");	goto Error; }
	cudaStatus = hipMemcpy(dev_a, mul_a, M * N * sizeof(T_type), hipMemcpyHostToDevice);	if (cudaStatus != hipSuccess){ fprintf(stderr, "cudamemcpy dev_a failed!\n");	goto Error; }
	cudaStatus = hipMemcpy(dev_b, mul_b, N * S * sizeof(T_type), hipMemcpyHostToDevice);	if (cudaStatus != hipSuccess){ fprintf(stderr, "hipMemcpy dev_b failed!\n"); goto Error; }
	hipEvent_t gpuStart, gpuFinish;  float elapsedTime;	hipEventCreate(&gpuStart);	hipEventCreate(&gpuFinish);hipEventRecord(gpuStart, 0);

	/*const int THREADNUM = 256; const int BLOCKNUM = (M * S + 255) / 256;*/ 

	const int BLOCK_SIZE = threadPerBlock;
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid((S + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);
	// gpuMatMultKernel << <grid, block >> >(dev_a, dev_b, dev_result, M, N, S);
	gpuMatMultWithSharedKernel<threadPerBlock, T_type> << <grid, block >> >(dev_a, dev_b, dev_result, M, N, S);

	hipEventRecord(gpuFinish, 0);
	hipEventSynchronize(gpuFinish);
	hipEventElapsedTime(&elapsedTime, gpuStart, gpuFinish);
	printf("\nThe runing time of GPU on Mat Multiply using sharing memory is %f seconds.\n", elapsedTime / 1000.0);
	cudaStatus = hipGetLastError(); if (cudaStatus != hipSuccess)	{fprintf(stderr, "MulKernel launch failed: %s!\n", hipGetErrorString(cudaStatus));	goto Error;	}
	// cudaStatus = hipDeviceSynchronize(); if (cudaStatus != hipSuccess){fprintf(stderr, "hipDeviceSynchronize return Error code %d after Kernel launched!\n", cudaStatus);goto Error; }
	cudaStatus = hipMemcpy(result, dev_result, M * S * sizeof(T_type), hipMemcpyDeviceToHost); if (cudaStatus != hipSuccess){ fprintf(stderr, "hipMemcpy result failed!\n"); goto Error; }
	
Error:
	hipUnbindTexture(texA);
	hipUnbindTexture(texB);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_result);

	return cudaStatus;
}

//调用CUDA运行GPU矩阵乘法核函数
//将矩阵A与矩阵B绑定到纹理内存中
hipError_t mulWithCudaTex(const int *mul_a, const int *mul_b, int *result, const int M, const int N, const int S)
{
	int * dev_a = 0;
	int * dev_b = 0;
	int * dev_result = 0;

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed! Do you have mul_a CUDA_capable GPU installed?\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_a, M * N * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_a failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_b, N * S * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_b failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_result, M * S * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_result failed!\n");
		goto Error;
	}

	hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
	cudaStatus = hipBindTexture(NULL, texA, dev_a, desc, M * N * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipBindTexture texA failed!\n");
		goto Error;
	}

	cudaStatus = hipBindTexture(NULL, texB, dev_b, desc, N * S * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipBindTexture texB failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_a, mul_a, M * N * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudamemcpy dev_a failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, mul_b, N * S * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy dev_b failed!\n");
		goto Error;
	}

	hipEvent_t gpuStart, gpuFinish;
	float elapsedTime;
	hipEventCreate(&gpuStart);
	hipEventCreate(&gpuFinish);
	hipEventRecord(gpuStart, 0);

	const int BLOCK_SIZE = 16;
	if ((M % BLOCK_SIZE != 0) && (S % BLOCK_SIZE != 0))
	{
		fprintf(stderr, "M or S can't be dividen by 16!\n");
		goto Error;
	}

	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(S / BLOCK_SIZE, M / BLOCK_SIZE);
	gpuMatMultWithTextureKernel << <grid, block >> >(dev_result, M, N, S);

	hipEventRecord(gpuFinish, 0);
	hipEventSynchronize(gpuFinish);
	hipEventElapsedTime(&elapsedTime, gpuStart, gpuFinish);
	printf("\nThe runing time of GPU on Mat Multiply using texture is %f seconds.\n", elapsedTime / 1000.0);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "MulKernel launch failed: %s!\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize return Error code %d after Kernel launched!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(result, dev_result, M * S * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy result failed!\n");
		goto Error;
	}

Error:
	hipUnbindTexture(texA);
	hipUnbindTexture(texB);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_result);

	return cudaStatus;

}
